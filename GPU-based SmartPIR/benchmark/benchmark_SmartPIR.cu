// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0
// Developer: Alişah Özcan

#include "heongpu.cuh"

#include <string>
#include <iomanip>
#include <omp.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include <random>

int main(int argc, char* argv[])
{
    hipSetDevice(0);

    std::size_t poly_modulus_degrees = 8192;
    std::size_t plain_modulus = 65537;

    heongpu::Parameters context(
        heongpu::scheme_type::bfv,
        heongpu::keyswitching_type::KEYSWITCHING_METHOD_I);
    context.set_poly_modulus_degree(poly_modulus_degrees);
    context.set_default_coeff_modulus(1);
    context.set_plain_modulus(plain_modulus);
    context.generate();

    heongpu::HEKeyGenerator keygen(context);
    heongpu::Secretkey secret_key(context);
    keygen.generate_secret_key(secret_key);

    heongpu::Publickey public_key(context);
    keygen.generate_public_key(public_key, secret_key);

    heongpu::Relinkey relin_key(context);
    keygen.generate_relin_key(relin_key, secret_key);

    std::vector<int> custom_key_index = {1};
    heongpu::Galoiskey galois_key(context, custom_key_index);
    keygen.generate_galois_key(galois_key, secret_key);

    heongpu::HEEncoder encoder(context);
    heongpu::HEEncryptor encryptor(context, public_key);
    heongpu::HEDecryptor decryptor(context, secret_key);
    heongpu::HEArithmeticOperator operators(context, encoder);

    // Easily, e assume the database contain N key-value items
    // assume each entry is 64 bits, num = 8192.
    
    heongpu::HostVector<heongpu::Plaintext> Value(4);
    
    for (int i = 0; i < 4; i++) {
        heongpu::HostVector<uint64_t> messagev(poly_modulus_degrees, 1);
        encoder.encode(Value[i], messagev);
    }
    
    heongpu::HostVector<uint64_t> query(poly_modulus_degrees, 1);
    query[0] = 1;
    heongpu::HostVector<heongpu::Plaintext> pQ;
    heongpu::HostVector<heongpu::Plaintext> cQ;
    encoder.encode(pQ,query);
        encryptor.encrypt(cQ, pQ);

    std::cout << "INFO: The server use the I to extract the target value." << std::endl;
    heongpu::HostVector<heongpu::Ciphertext> cV(4);
    for (int i = 0; i < 4; i++) {
        operators.multiply_plain(cQ, Value[i], cV[i]);
    }
    heongpu::Ciphertext Ans(context);
    for (int i = 0; i < t; i++) {
        operators.rotate_rows(cV[i], cV[i], galois_key, 1);
        operators.add(Ans, cV[i], Ans);
    }
    
    return EXIT_SUCCESS;
}