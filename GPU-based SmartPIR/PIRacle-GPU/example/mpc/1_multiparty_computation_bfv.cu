// Copyright 2024 Alişah Özcan
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0
// Developer: Alişah Özcan

#include "heongpu.cuh"
#include "../example_util.h"
#include <omp.h>

int main(int argc, char* argv[])
{
    hipSetDevice(0);

    heongpu::Parameters context(
        heongpu::scheme_type::bfv,
        heongpu::keyswitching_type::KEYSWITCHING_METHOD_I);

    size_t poly_modulus_degree = 8192;
    context.set_poly_modulus_degree(poly_modulus_degree);
    context.set_default_coeff_modulus(1);
    int plain_modulus = 1032193;
    context.set_plain_modulus(plain_modulus);
    context.generate();
    context.print_parameters();

    std::random_device rd;
    std::mt19937 gen(rd());
    int common_seed = gen();
    std::cout << "Common seed: " << common_seed << std::endl;

    std::vector<int> shift_value = {1};

    ///////////////////////////////////////////////////////////
    ///////////// Alice Setup (Stage 1) (Phases 1) ////////////
    ///////////////////////////////////////////////////////////

    heongpu::HEKeyGenerator keygen_alice(context);
    heongpu::Secretkey secret_key_alice(context);
    keygen_alice.generate_secret_key(secret_key_alice);

    // Publickey
    heongpu::MultipartyPublickey public_key_alice(context, common_seed);
    keygen_alice.generate_multi_party_public_key_piece(public_key_alice,
                                                       secret_key_alice);

    // Relinkey
    heongpu::MultipartyRelinkey relin_key_alice_stage1(context, common_seed);
    keygen_alice.generate_multi_party_relin_key_piece(relin_key_alice_stage1,
                                                      secret_key_alice);

    // Galoiskey
    heongpu::MultipartyGaloiskey galois_key_alice(context, shift_value,
                                                  common_seed);
    keygen_alice.generate_multi_party_galios_key_piece(galois_key_alice,
                                                       secret_key_alice);

    ///////////////////////////////////////////////////////////
    ////////////// Bob Setup (Stage 1) (Phases 1) /////////////
    ///////////////////////////////////////////////////////////

    heongpu::HEKeyGenerator keygen_bob(context);
    heongpu::Secretkey secret_key_bob(context);
    keygen_bob.generate_secret_key(secret_key_bob);

    // Publickey
    heongpu::MultipartyPublickey public_key_bob(context, common_seed);
    keygen_bob.generate_multi_party_public_key_piece(public_key_bob,
                                                     secret_key_bob);

    // Relinkey
    heongpu::MultipartyRelinkey relin_key_bob_stage1(context, common_seed);
    keygen_bob.generate_multi_party_relin_key_piece(relin_key_bob_stage1,
                                                    secret_key_bob);

    // Galoiskey
    heongpu::MultipartyGaloiskey galois_key_bob(context, shift_value,
                                                common_seed);
    keygen_bob.generate_multi_party_galios_key_piece(galois_key_bob,
                                                     secret_key_bob);

    ///////////////////////////////////////////////////////////
    /////////// Charlie Setup (Stage 1) (Phases 1) ////////////
    ///////////////////////////////////////////////////////////

    heongpu::HEKeyGenerator keygen_charlie(context);
    heongpu::Secretkey secret_key_charlie(context);
    keygen_charlie.generate_secret_key(secret_key_charlie);

    // Publickey
    heongpu::MultipartyPublickey public_key_charlie(context, common_seed);
    keygen_charlie.generate_multi_party_public_key_piece(public_key_charlie,
                                                         secret_key_charlie);

    // Relinkey
    heongpu::MultipartyRelinkey relin_key_charlie_stage1(context, common_seed);
    keygen_charlie.generate_multi_party_relin_key_piece(
        relin_key_charlie_stage1, secret_key_charlie);

    // Galoiskey
    heongpu::MultipartyGaloiskey galois_key_charlie(context, shift_value,
                                                    common_seed);
    keygen_charlie.generate_multi_party_galios_key_piece(galois_key_charlie,
                                                         secret_key_charlie);

    ///////////////////////////////////////////////////////////
    ///////////// Key Sharing (Stage 1) (Phases 1) ////////////
    ///////////////////////////////////////////////////////////

    std::vector<heongpu::MultipartyPublickey> participant_public_keys;
    participant_public_keys.push_back(public_key_alice);
    participant_public_keys.push_back(public_key_bob);
    participant_public_keys.push_back(public_key_charlie);

    std::vector<heongpu::MultipartyRelinkey> participant_relin_keys_stage1;
    participant_relin_keys_stage1.push_back(relin_key_alice_stage1);
    participant_relin_keys_stage1.push_back(relin_key_bob_stage1);
    participant_relin_keys_stage1.push_back(relin_key_charlie_stage1);

    std::vector<heongpu::MultipartyGaloiskey> participant_galois_keys;
    participant_galois_keys.push_back(galois_key_alice);
    participant_galois_keys.push_back(galois_key_bob);
    participant_galois_keys.push_back(galois_key_charlie);

    heongpu::HEKeyGenerator keygen_server(context);
    heongpu::Publickey common_public_key(context);
    keygen_server.generate_multi_party_public_key(participant_public_keys,
                                                  common_public_key);

    heongpu::MultipartyRelinkey common_relin_key_stage1(context, common_seed);
    keygen_server.generate_multi_party_relin_key(participant_relin_keys_stage1,
                                                 common_relin_key_stage1);

    heongpu::Galoiskey common_galois_key(context, shift_value);
    keygen_server.generate_multi_party_galois_key(participant_galois_keys,
                                                  common_galois_key);

    ///////////////////////////////////////////////////////////
    ///////////// Alice Setup (Stage 1) (Phases 2) ////////////
    ///////////////////////////////////////////////////////////

    // Relinkey
    heongpu::MultipartyRelinkey relin_key_alice_stage2(context, common_seed);
    keygen_alice.generate_multi_party_relin_key_piece(
        common_relin_key_stage1, relin_key_alice_stage2, secret_key_alice);

    ///////////////////////////////////////////////////////////
    ////////////// Bob Setup (Stage 1) (Phases 2) /////////////
    ///////////////////////////////////////////////////////////

    // Relinkey
    heongpu::MultipartyRelinkey relin_key_bob_stage2(context, common_seed);
    keygen_bob.generate_multi_party_relin_key_piece(
        common_relin_key_stage1, relin_key_bob_stage2, secret_key_bob);

    ///////////////////////////////////////////////////////////
    //////////// Charlie Setup (Stage 1) (Phases 2) ///////////
    ///////////////////////////////////////////////////////////

    // Relinkey
    heongpu::MultipartyRelinkey relin_key_charlie_stage2(context, common_seed);
    keygen_charlie.generate_multi_party_relin_key_piece(
        common_relin_key_stage1, relin_key_charlie_stage2, secret_key_charlie);

    ///////////////////////////////////////////////////////////
    //////////// Key Sharing (Stage 1) (Phases 2) /////////////
    ///////////////////////////////////////////////////////////

    std::vector<heongpu::MultipartyRelinkey> participant_relin_keys_stage2;
    participant_relin_keys_stage2.push_back(relin_key_alice_stage2);
    participant_relin_keys_stage2.push_back(relin_key_bob_stage2);
    participant_relin_keys_stage2.push_back(relin_key_charlie_stage2);

    heongpu::Relinkey common_relin_key(context, common_seed);
    keygen_server.generate_multi_party_relin_key(participant_relin_keys_stage2,
                                                 common_relin_key_stage1,
                                                 common_relin_key);

    ///////////////////////////////////////////////////////////
    ////////////////// Alice Setup (Stage 2) //////////////////
    ///////////////////////////////////////////////////////////

    heongpu::HEEncoder encoder_alice(context);
    heongpu::HEEncryptor encryptor_alice(context, common_public_key);

    // Generate simple matrix in CPU.
    const int row_size = poly_modulus_degree / 2;
    std::vector<uint64_t> message_alice(poly_modulus_degree, 7ULL); // In CPU
    message_alice[0] = 1ULL;
    message_alice[1] = 10ULL;
    message_alice[2] = 100ULL;

    display_matrix(message_alice, row_size);

    heongpu::Plaintext plaintext_alice(context);
    encoder_alice.encode(plaintext_alice, message_alice);

    heongpu::Ciphertext ciphertext_alice(context);
    encryptor_alice.encrypt(ciphertext_alice, plaintext_alice);

    ///////////////////////////////////////////////////////////
    /////////////////// Bob Setup (Stage 2) ///////////////////
    ///////////////////////////////////////////////////////////

    heongpu::HEEncoder encoder_bob(context);
    heongpu::HEEncryptor encryptor_bob(context, common_public_key);

    // Generate simple matrix in CPU.
    std::vector<uint64_t> message_bob(poly_modulus_degree, 8ULL); // In CPU
    message_bob[0] = 2ULL;
    message_bob[1] = 20ULL;
    message_bob[2] = 200ULL;

    display_matrix(message_bob, row_size);

    heongpu::Plaintext plaintext_bob(context);
    encoder_bob.encode(plaintext_bob, message_bob);

    heongpu::Ciphertext ciphertext_bob(context);
    encryptor_bob.encrypt(ciphertext_bob, plaintext_bob);

    ///////////////////////////////////////////////////////////
    ///////////////// Charlie Setup (Stage 2) /////////////////
    ///////////////////////////////////////////////////////////

    heongpu::HEEncoder encoder_charlie(context);
    heongpu::HEEncryptor encryptor_charlie(context, common_public_key);

    // Generate simple matrix in CPU.
    std::vector<uint64_t> message_charlie(poly_modulus_degree, 9ULL); // In CPU
    message_charlie[0] = 3ULL;
    message_charlie[1] = 30ULL;
    message_charlie[2] = 300ULL;

    display_matrix(message_charlie, row_size);

    heongpu::Plaintext plaintext_charlie(context);
    encoder_charlie.encode(plaintext_charlie, message_charlie);

    heongpu::Ciphertext ciphertext_charlie(context);
    encryptor_charlie.encrypt(ciphertext_charlie, plaintext_charlie);

    ///////////////////////////////////////////////////////////
    ///////////////// Server Setup (Stage 3) //////////////////
    ///////////////////////////////////////////////////////////

    heongpu::HEArithmeticOperator operators(context, encoder_charlie);

    heongpu::Ciphertext cipher_mult(context);
    operators.multiply(ciphertext_alice, ciphertext_bob, cipher_mult);
    operators.relinearize_inplace(cipher_mult, common_relin_key);

    heongpu::Ciphertext cipher_mult_add(context);
    operators.add(cipher_mult, ciphertext_charlie, cipher_mult_add);

    heongpu::Ciphertext cipher_mult_add_rotate(context);
    operators.rotate_rows(cipher_mult_add, cipher_mult_add_rotate,
                          common_galois_key, 1);

    ///////////////////////////////////////////////////////////
    /////////////////// Alice Setup (Stage 4) /////////////////
    ///////////////////////////////////////////////////////////

    heongpu::HEDecryptor decryptor_alice(context, secret_key_alice);

    heongpu::Ciphertext partial_ciphertext_alice(context);
    decryptor_alice.multi_party_decrypt_partial(
        cipher_mult_add_rotate, secret_key_alice, partial_ciphertext_alice);

    ///////////////////////////////////////////////////////////
    /////////////////// Bob Setup (Stage 4) ///////////////////
    ///////////////////////////////////////////////////////////

    heongpu::HEDecryptor decryptor_bob(context, secret_key_alice);

    heongpu::Ciphertext partial_ciphertext_bob(context);
    decryptor_bob.multi_party_decrypt_partial(
        cipher_mult_add_rotate, secret_key_bob, partial_ciphertext_bob);

    ///////////////////////////////////////////////////////////
    ///////////////// Charlie Setup (Stage 4) /////////////////
    ///////////////////////////////////////////////////////////

    heongpu::HEDecryptor decryptor_charlie(context, secret_key_alice);

    heongpu::Ciphertext partial_ciphertext_charlie(context);
    decryptor_charlie.multi_party_decrypt_partial(
        cipher_mult_add_rotate, secret_key_charlie, partial_ciphertext_charlie);

    ///////////////////////////////////////////////////////////

    std::vector<heongpu::Ciphertext> partial_ciphertexts;
    partial_ciphertexts.push_back(partial_ciphertext_alice);
    partial_ciphertexts.push_back(partial_ciphertext_bob);
    partial_ciphertexts.push_back(partial_ciphertext_charlie);

    heongpu::Plaintext plaintext_result(context);
    decryptor_alice.multi_party_decrypt_fusion(partial_ciphertexts,
                                               plaintext_result);

    std::vector<uint64_t> check_result;
    encoder_alice.decode(check_result, plaintext_result);

    display_matrix(check_result, row_size);

    return EXIT_SUCCESS;
}
