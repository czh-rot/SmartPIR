#include "hip/hip_runtime.h"
// Copyright 2024 Alişah Özcan
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0
// Developer: Alişah Özcan

#include "heongpu.cuh"
#include "../example_util.h"

int main(int argc, char* argv[])
{
    hipSetDevice(0); // Use it for memory pool

    // Initialize encryption parameters for the CKKS scheme.
    heongpu::Parameters context(
        heongpu::scheme_type::ckks,
        heongpu::keyswitching_type::KEYSWITCHING_METHOD_II,
        heongpu::sec_level_type::none);
    size_t poly_modulus_degree = 4096;
    context.set_poly_modulus_degree(poly_modulus_degree);

    // Last modulus has to be three times the value of scale, otherwise, it will
    // fail. 3298535538689ULL =~ 3 * pow(2.0, 40)
    context.set_custom_coeff_modulus(
        {3298535538689ULL, 1099512938497ULL, 1099515691009ULL,
         1099516870657ULL, 1099521458177ULL, 1099522375681ULL,
         1099523555329ULL, 1099525128193ULL, 1099526176769ULL,
         1099529060353ULL, 1099535220737ULL, 1099536138241ULL,
         1099537580033ULL, 1099538104321ULL, 1099540725761ULL,
         1099540856833ULL, 1099543085057ULL, 1099544002561ULL,
         1099544395777ULL, 1099548327937ULL},
        {3298535669761ULL, 3298538684417ULL, 3298540650497ULL});
    context.generate();
    context.print_parameters();

    // The scale is set to 2^40, resulting in 40 bits of precision before the
    // decimal point.
    double scale = pow(2.0, 40);

    // Generate keys: the public key for encryption, the secret key for
    // decryption and evaluation key(relinkey) for relinearization.
    heongpu::HEKeyGenerator keygen(context);
    heongpu::Secretkey secret_key(context,
                                  16); // hamming weight is 16 in this example
    keygen.generate_secret_key(secret_key);

    heongpu::Publickey public_key(context);
    keygen.generate_public_key(public_key, secret_key);

    heongpu::Relinkey relin_key(context);
    keygen.generate_relin_key(relin_key, secret_key);

    // Initialize Encoder, Encryptor, Evaluator, and Decryptor. The Encoder will
    // encode the message for SIMD operations. The Encryptor will use the public
    // key to encrypt data, while the Decryptor will use the secret key to
    // decrypt it. The Evaluator will handle operations on the encrypted data.
    heongpu::HEEncoder encoder(context);
    heongpu::HEEncryptor encryptor(context, public_key);
    heongpu::HEDecryptor decryptor(context, secret_key);
    // heongpu::HEOperator operators(context);
    heongpu::HELogicOperator operators(context, encoder, scale);

    // Generate simple vector in CPU.
    const int slot_count = poly_modulus_degree / 2;
    std::cout << "Plaintext vector size: " << slot_count << std::endl;
    std::vector<double> message1(slot_count, 1);
    message1[0] = 0;
    message1[2] = 0;

    std::vector<double> message2(slot_count, 1);

    //  Transfer that vector from CPU to GPU and Encode that simple vector in
    //  GPU.
    heongpu::Plaintext P1(context);
    encoder.encode(P1, message1, scale);

    heongpu::Plaintext P2(context);
    encoder.encode(P2, message2, scale);

    heongpu::Ciphertext C1(context);
    encryptor.encrypt(C1, P1);

    heongpu::Ciphertext C2(context);
    encryptor.encrypt(C2, P2);

    // Check README.md for more detail information
    // CtoS_piece_ = [2,5]
    // StoC_piece_ = [2,5]
    // taylor_number_ = [6,15]
    // less_key_mode_ = true or false
    int StoC_piece = 3;
    heongpu::BootstrappingConfig boot_config(3, StoC_piece, 6, true);
    // Generates all bootstrapping parameters before bootstrapping
    // operators.generate_bootstrapping_parameters(encoder, scale, boot_config);
    operators.generate_bootstrapping_params(
        scale, boot_config,
        heongpu::logic_bootstrapping_type::GATE_BOOTSTRAPPING);

    std::vector<int> key_index = operators.bootstrapping_key_indexs();
    std::cout << "Total galois key needed for CKKS bootstrapping: "
              << key_index.size() << std::endl;
    heongpu::Galoiskey galois_key(
        context, key_index); // all galois keys are stored in GPU
    // heongpu::Galoiskey galois_key(context,key_index, false); // all galois
    // keys are stored in CPU

    // Generates all galois key needed for bootstrapping
    keygen.generate_galois_key(galois_key, secret_key);

    // Drop all level until one level remain
    for (int i = 0; i < (19 - StoC_piece); i++)
    {
        operators.mod_drop_inplace(C1);
        operators.mod_drop_inplace(C2);
    }

    std::cout << "Depth before bootstrapping: " << C1.depth() << std::endl;

    // Bootstapping Operation
    heongpu::Ciphertext cipher_boot =
        operators.AND_bootstrapping(C1, C2, galois_key, relin_key);

    std::cout << "Depth after bootstrapping: " << cipher_boot.depth()
              << std::endl;

    heongpu::Plaintext P_res1(context);
    decryptor.decrypt(P_res1, cipher_boot);
    std::vector<Complex64> decrypted_1;
    encoder.decode(decrypted_1, P_res1);

    // for(int j = 0; j < slot_count; j++){
    for (int j = 0; j < 16; j++)
    {
        std::cout << j
                  << "-> EXPECTED:" << (int(message1[j]) & int(message1[j]))
                  << " - ACTUAL:" << decrypted_1[j] << std::endl;
    }
    std::cout << std::endl;

    return EXIT_SUCCESS;
}
