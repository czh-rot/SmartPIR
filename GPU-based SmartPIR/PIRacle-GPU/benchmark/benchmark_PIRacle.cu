// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0
// Developer: Alişah Özcan

#include "heongpu.cuh"

#include <string>
#include <iomanip>
#include <omp.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include <random>

std::vector<std::vector<int>> generateConstantWeightVectors(int N, int m, int w) {
    std::vector<std::vector<int>> vectors(N, std::vector<int>(m, 0));  // ✅ 预分配
    std::random_device rd;  
    std::mt19937 gen(rd());

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < w; ++j) {
            vectors[i][j] = 1;  
        }
        std::shuffle(vectors[i].begin(), vectors[i].end(), gen);
    }
    
    return vectors;
}

int main(int argc, char* argv[])
{
    hipSetDevice(0);

    std::size_t poly_modulus_degrees = 32768;
    std::size_t plain_modulus = 65537;

    heongpu::Parameters context(
        heongpu::scheme_type::bfv,
        heongpu::keyswitching_type::KEYSWITCHING_METHOD_I);
    context.set_poly_modulus_degree(poly_modulus_degrees);
    context.set_default_coeff_modulus(1);
    context.set_plain_modulus(plain_modulus);
    context.generate();

    heongpu::HEKeyGenerator keygen(context);
    heongpu::Secretkey secret_key(context);
    keygen.generate_secret_key(secret_key);

    heongpu::Publickey public_key(context);
    keygen.generate_public_key(public_key, secret_key);

    heongpu::Relinkey relin_key(context);
    keygen.generate_relin_key(relin_key, secret_key);

    std::vector<int> custom_key_index = {1};
    heongpu::Galoiskey galois_key(context, custom_key_index);
    keygen.generate_galois_key(galois_key, secret_key);

    heongpu::HEEncoder encoder(context);
    heongpu::HEEncryptor encryptor(context, public_key);
    heongpu::HEDecryptor decryptor(context, secret_key);
    heongpu::HEArithmeticOperator operators(context, encoder);

    // Easily, e assume the database contain N key-value items
    int m = 60;
    int w = 30;
    int t = 40;
    int n = poly_modulus_degrees;
    int dim1 = n / poly_modulus_degrees;
    heongpu::HostVector<heongpu::Plaintext> Key(dim1 * m);
    heongpu::HostVector<heongpu::Plaintext> Value(t);
    auto mm = generateConstantWeightVectors(n, m, w); //m[n][m]

    heongpu::HostVector<uint64_t> messagek(poly_modulus_degrees, 0);
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            messagek[j] = static_cast<uint64_t>(mm[i][j]);
        }
        encoder.encode(Key[i], messagek);
    }
    
    for (int i = 0; i < t; i++) {
        heongpu::HostVector<uint64_t> messagev(poly_modulus_degrees, 1);
        encoder.encode(Value[i], messagev);
    }
    
    // Client: we assume the client wanna query the cwc[0];
    std::cout << "INFO: The client encrypts the key 0 and sends it to the server." << std::endl;
    heongpu::HostVector<heongpu::Plaintext> pQ(m);
    heongpu::HostVector<heongpu::Ciphertext> cQ(m); // encrypted query
    for (int i = 0; i < m; i++) {
        heongpu::HostVector<uint64_t> temp(poly_modulus_degrees, 0);
        fill(temp.begin(), temp.end(), mm[0][i]);
        encoder.encode(pQ[i], temp);
        encryptor.encrypt(cQ[i], pQ[i]);
    }

    std::cout << "INFO: The server receives the encrypted query, and compute the I." << std::endl;
    heongpu::HostVector<heongpu::Ciphertext> res(m);
    for (int i = 0; i < m; i++) {
        operators.multiply_plain_inplace(cQ[i], Key[i]);
    }
    heongpu::Ciphertext sum(context);
    for (int i = 0; i < m; i++) {
        operators.add_inplace(sum, cQ[i]);
    }
    heongpu::Ciphertext One(context);
    heongpu::Ciphertext W(context);
    heongpu::Ciphertext Dif(context);
    heongpu::Plaintext p_One(context);
    heongpu::Plaintext p_W(context);
    heongpu::HostVector<uint64_t> cone(poly_modulus_degrees, 1);
    heongpu::HostVector<uint64_t> cw(poly_modulus_degrees, w);
    encoder.encode(p_One, cone);
    encoder.encode(p_W, cw);
    encryptor.encrypt(One, p_One);
    encryptor.encrypt(W, p_W);
    operators.sub(One, W, Dif);
    
    for (int i = 0; i < 16; i++) {
        operators.multiply_inplace(Dif, Dif);
        operators.relinearize_inplace(Dif, relin_key);
    }
    operators.sub(Dif, One, Dif);

    std::cout << "INFO: The server use the I to extract the target value." << std::endl;
    heongpu::HostVector<heongpu::Ciphertext> cV(t);
    for (int i = 0; i < t; i++) {
        operators.multiply_plain(Dif, Value[i], cV[i]);
    }
    heongpu::Ciphertext Ans(context);
    for (int i = 0; i < t; i++) {
        operators.rotate_rows(cV[i], cV[i], galois_key, 1);
        operators.add(Ans, cV[i], Ans);
    }
    
    return EXIT_SUCCESS;
}