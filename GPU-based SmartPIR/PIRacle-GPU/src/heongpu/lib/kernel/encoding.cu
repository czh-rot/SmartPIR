#include "hip/hip_runtime.h"
﻿// Copyright 2024 Alişah Özcan
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0
// Developer: Alişah Özcan

#include "encoding.cuh"

namespace heongpu
{

    __global__ void encode_kernel_bfv(Data64* message_encoded, Data64* message,
                                      Data64* location_info,
                                      Modulus64* plain_mod, int message_size)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        int location = location_info[idx];

        if (idx < message_size)
        {
            int64_t message_in = static_cast<int64_t>(message[idx]);
            message_in =
                (message_in < 0) ? message_in + plain_mod[0].value : message_in;

            message_encoded[location] = static_cast<Data64>(message_in);
        }
        else
        {
            Data64 zero = 0;
            message_encoded[location] = zero;
        }
    }

    __global__ void decode_kernel_bfv(Data64* message, Data64* message_encoded,
                                      Data64* location_info)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        int location = location_info[idx];
        message[idx] = message_encoded[location];
    }

    __global__ void encode_kernel_double_ckks_conversion(
        Data64* plaintext, double message, Modulus64* modulus,
        int coeff_modulus_count, double two_pow_64, int n_power)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x; // ring_size

        double message_r = message;

        double coeff_double = round(message_r);
        bool is_negative = signbit(coeff_double);
        coeff_double = fabs(coeff_double);

        // Change Type
        Data64 coeff[2] = {
            static_cast<std::uint64_t>(fmod(coeff_double, two_pow_64)),
            static_cast<std::uint64_t>(coeff_double / two_pow_64)};

        if (is_negative)
        {
            for (int i = 0; i < coeff_modulus_count; i++)
            {
                Data64 temp = OPERATOR_GPU_64::reduce(coeff, modulus[i]);
                plaintext[idx + (i << n_power)] =
                    OPERATOR_GPU_64::sub(modulus[i].value, temp, modulus[i]);
            }
        }
        else
        {
            for (int i = 0; i < coeff_modulus_count; i++)
            {
                plaintext[idx + (i << n_power)] =
                    OPERATOR_GPU_64::reduce(coeff, modulus[i]);
            }
        }
    }

    __global__ void encode_kernel_int_ckks_conversion(Data64* plaintext,
                                                      std::int64_t message,
                                                      Modulus64* modulus,
                                                      int n_power)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x; // ring_size
        int block_y = blockIdx.y;
        int location = idx + (block_y << n_power);

        Modulus64 mod = modulus[block_y];
        std::int64_t message_r = message;

        if (message < 0)
        {
            message_r = message_r + mod.value;
            Data64 message_d = static_cast<Data64>(message_r);
            message_d = OPERATOR_GPU_64::reduce_forced(message_d, mod);
            plaintext[location] = message_d;
        }
        else
        {
            Data64 message_d = static_cast<Data64>(message_r);
            message_d = OPERATOR_GPU_64::reduce_forced(message_d, mod);
            plaintext[location] = message_d;
        }
    }

    //////////////////////////////
    //////////////////////////////
    //////////////////////////////

    __global__ void double_to_complex_kernel(double* input, Complex64* output)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        double in = input[idx];

        Complex64 c_in(in, 0.0);
        output[idx] = c_in;
    }

    __global__ void complex_to_double_kernel(Complex64* input, double* output)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        Complex64 in = input[idx];

        double d_in = in.real();
        output[idx] = d_in;
    }

    //

    __global__ void
    encode_kernel_ckks_conversion(Data64* plaintext, Complex64* complex_message,
                                  Modulus64* modulus, int coeff_modulus_count,
                                  double two_pow_64, int* reverse_order,
                                  int n_power)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x; // slot_count

        int order = reverse_order[idx];
        Complex64 partial_message = complex_message[order];

        double coeff_double = round(partial_message.real());
        bool is_negative = signbit(coeff_double);
        coeff_double = fabs(coeff_double);

        // Change Type
        Data64 coeff[2] = {
            static_cast<std::uint64_t>(fmod(coeff_double, two_pow_64)),
            static_cast<std::uint64_t>(coeff_double / two_pow_64)};

        if (is_negative)
        {
            for (int i = 0; i < coeff_modulus_count; i++)
            {
                Data64 temp = OPERATOR_GPU_64::reduce(coeff, modulus[i]);
                plaintext[idx + (i << n_power)] =
                    OPERATOR_GPU_64::sub(modulus[i].value, temp, modulus[i]);
            }
        }
        else
        {
            for (int i = 0; i < coeff_modulus_count; i++)
            {
                plaintext[idx + (i << n_power)] =
                    OPERATOR_GPU_64::reduce(coeff, modulus[i]);
            }
        }

        // TODO: make it efficient
        int offset = 1 << (n_power - 1);

        double coeff_double2 = round(partial_message.imag());
        bool is_negative2 = signbit(coeff_double2);
        coeff_double2 = fabs(coeff_double2);

        // Change Type
        Data64 coeff2[2] = {
            static_cast<std::uint64_t>(fmod(coeff_double2, two_pow_64)),
            static_cast<std::uint64_t>(coeff_double2 / two_pow_64)};

        if (is_negative2)
        {
            for (int i = 0; i < coeff_modulus_count; i++)
            {
                Data64 temp = OPERATOR_GPU_64::reduce(coeff2, modulus[i]);
                plaintext[idx + offset + (i << n_power)] =
                    OPERATOR_GPU_64::sub(modulus[i].value, temp, modulus[i]);
            }
        }
        else
        {
            for (int i = 0; i < coeff_modulus_count; i++)
            {
                plaintext[idx + offset + (i << n_power)] =
                    OPERATOR_GPU_64::reduce(coeff2, modulus[i]);
            }
        }
    }

    __global__ void encode_kernel_compose(
        Complex64* complex_message, Data64* plaintext, Modulus64* modulus,
        Data64* Mi_inv, Data64* Mi, Data64* upper_half_threshold,
        Data64* decryption_modulus, int coeff_modulus_count, double scale,
        double two_pow_64, int* reverse_order, int n_power)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x; // slot_count
        double inv_scale = double(1.0) / scale;
        double two_pow_64_reg = two_pow_64;
        int offset = 1 << (n_power - 1);

        Data64 compose_result[50]; // TODO: Define size as global variable
        Data64 big_integer_result[50]; // TODO: Define size as global variable

        biginteger::set_zero(compose_result, coeff_modulus_count);

#pragma unroll
        for (int i = 0; i < coeff_modulus_count; i++)
        {
            Data64 base = plaintext[idx + (i << n_power)];
            Data64 temp = OPERATOR_GPU_64::mult(base, Mi_inv[i], modulus[i]);

            biginteger::multiply(Mi + (i * coeff_modulus_count),
                                 coeff_modulus_count, temp, big_integer_result,
                                 coeff_modulus_count);

            int carry = biginteger::add_inplace(
                compose_result, big_integer_result, coeff_modulus_count);

            bool check = biginteger::is_greater_or_equal(
                compose_result, decryption_modulus, coeff_modulus_count);

            if (check)
            {
                biginteger::sub2(compose_result, decryption_modulus,
                                 coeff_modulus_count, compose_result);
            }
        }

        double result_real = double(0.0);

        bool check1 = biginteger::is_greater_or_equal(
            compose_result, upper_half_threshold, coeff_modulus_count);

        if (check1)
        {
            double scaled_two_pow_64 = inv_scale;
            for (std::size_t j = 0; j < coeff_modulus_count;
                 j++, scaled_two_pow_64 *= two_pow_64_reg)
            {
                if (compose_result[j] > decryption_modulus[j])
                {
                    auto diff = compose_result[j] - decryption_modulus[j];
                    result_real +=
                        diff ? static_cast<double>(diff) * scaled_two_pow_64
                             : 0.0;
                }
                else
                {
                    auto diff = decryption_modulus[j] - compose_result[j];
                    result_real -=
                        diff ? static_cast<double>(diff) * scaled_two_pow_64
                             : 0.0;
                }
            }
        }
        else
        {
            double scaled_two_pow_64 = inv_scale;
            for (std::size_t j = 0; j < coeff_modulus_count;
                 j++, scaled_two_pow_64 *= two_pow_64_reg)
            {
                auto curr_coeff = compose_result[j];
                result_real += curr_coeff ? static_cast<double>(curr_coeff) *
                                                scaled_two_pow_64
                                          : 0.0;
            }
        }

        //////////////////////////
        //////////////////////////
        // TODO: make it efficient
        biginteger::set_zero(compose_result, coeff_modulus_count);

#pragma unroll
        for (int i = 0; i < coeff_modulus_count; i++)
        {
            Data64 base = plaintext[idx + offset + (i << n_power)];
            Data64 temp = OPERATOR_GPU_64::mult(base, Mi_inv[i], modulus[i]);

            biginteger::multiply(Mi + (i * coeff_modulus_count),
                                 coeff_modulus_count, temp, big_integer_result,
                                 coeff_modulus_count);

            int carry = biginteger::add_inplace(
                compose_result, big_integer_result, coeff_modulus_count);

            bool check = biginteger::is_greater_or_equal(
                compose_result, decryption_modulus, coeff_modulus_count);

            if (check)
            {
                biginteger::sub2(compose_result, decryption_modulus,
                                 coeff_modulus_count, compose_result);
            }
        }

        double result_imag = double(0.0);

        bool check2 = biginteger::is_greater_or_equal(
            compose_result, upper_half_threshold, coeff_modulus_count);

        if (check2)
        {
            double scaled_two_pow_64 = inv_scale;
            for (std::size_t j = 0; j < coeff_modulus_count;
                 j++, scaled_two_pow_64 *= two_pow_64_reg)
            {
                if (compose_result[j] > decryption_modulus[j])
                {
                    auto diff = compose_result[j] - decryption_modulus[j];
                    result_imag +=
                        diff ? static_cast<double>(diff) * scaled_two_pow_64
                             : 0.0;
                }
                else
                {
                    auto diff = decryption_modulus[j] - compose_result[j];
                    result_imag -=
                        diff ? static_cast<double>(diff) * scaled_two_pow_64
                             : 0.0;
                }
            }
        }
        else
        {
            double scaled_two_pow_64 = inv_scale;
            for (std::size_t j = 0; j < coeff_modulus_count;
                 j++, scaled_two_pow_64 *= two_pow_64_reg)
            {
                auto curr_coeff = compose_result[j];
                result_imag += curr_coeff ? static_cast<double>(curr_coeff) *
                                                scaled_two_pow_64
                                          : 0.0;
            }
        }

        Complex64 result_c(result_real, result_imag);

        int order = reverse_order[idx];
        complex_message[order] = result_c;
    }

} // namespace heongpu