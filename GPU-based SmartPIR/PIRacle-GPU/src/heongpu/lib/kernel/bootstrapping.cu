#include "hip/hip_runtime.h"
// Copyright 2024 Alişah Özcan
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0
// Developer: Alişah Özcan

#include "bootstrapping.cuh"

namespace heongpu
{

    __device__ int exponent_calculation(int& index, int& n)
    {
        Data64 result = 1ULL;
        Data64 five = 5ULL;
        Data64 mod = (n << 2) - 1;

        int bits = 32 - __clz(index);
        for (int i = bits - 1; i > -1; i--)
        {
            result = (result * result) & mod;

            if (((index >> i) & 1u))
            {
                result = (result * five) & mod;
            }
        }

        return result;
    }

    __device__ int matrix_location(int& index)
    {
        if (index == 0)
        {
            return 0;
        }

        return (3 * index) - 1;
    }

    __device__ int matrix_reverse_location(int& index)
    {
        int total = (gridDim.y - 1) * 3;
        if (index == 0)
        {
            return total;
        }

        return total - (3 * index);
    }

    __global__ void E_diagonal_generate_kernel(Complex64* output, int n_power)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        int block_y = blockIdx.y; // matrix index
        int logk = block_y + 1;
        int output_location = matrix_location(block_y);

        int n = 1 << n_power;
        int v_size = 1 << (n_power - logk);

        int index1 = idx & ((v_size << 1) - 1);
        int index2 = index1 >> (n_power - logk);
        Complex64 W1(1.0, 0.0);
        Complex64 W2(0.0, 0.0);
        Complex64 W3(0.0, 0.0);

        if (block_y == 0)
        {
            double angle = M_PI / (v_size << 2);
            Complex64 omega_4n(cos(angle), sin(angle));
            int expo = exponent_calculation(index1, n);

            Complex64 W = omega_4n.exp(expo);
            Complex64 W_neg = W; // W.negate();

            if (index2 == 1)
            {
                W1 = W_neg;
                W2 = Complex64(1.0, 0.0);
            }
            else
            {
                W2 = W;
            }

            output[(output_location << n_power) + idx] = W1;
            output[((output_location + 1) << n_power) + idx] = W2;
        }
        else
        {
            double angle = M_PI / (v_size << 2);
            Complex64 omega_4n(cos(angle), sin(angle));
            int expo = exponent_calculation(index1, n);

            Complex64 W = omega_4n.exp(expo);
            Complex64 W_neg = W; // W.negate();

            if (index2 == 1)
            {
                W1 = W_neg;
                W3 = Complex64(1.0, 0.0);
            }
            else
            {
                W2 = W;
            }

            output[(output_location << n_power) + idx] = W1;
            output[((output_location + 1) << n_power) + idx] = W2;
            output[((output_location + 2) << n_power) + idx] = W3;
        }
    }

    __global__ void E_diagonal_inverse_generate_kernel(Complex64* output,
                                                       int n_power)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        int block_y = blockIdx.y; // matrix index
        int logk = block_y + 1;
        int output_location = matrix_reverse_location(block_y);

        int n = 1 << n_power;
        int v_size = 1 << (n_power - logk);

        int index1 = idx & ((v_size << 1) - 1);
        int index2 = index1 >> (n_power - logk);
        Complex64 W1(0.5, 0.0);
        Complex64 W2(0.5, 0.0);
        Complex64 W3(0.0, 0.0);

        if (block_y == 0)
        {
            if (index2 == 1)
            {
                double angle = M_PI / (v_size << 2);
                Complex64 omega_4n(cos(angle), sin(angle));
                int expo = exponent_calculation(index1, n);
                W1 = omega_4n.inverse();
                W1 = W1.exp(expo);
                W1 = W1 / Complex64(2.0, 0.0);
                W2 = W1.negate();
            }

            output[(output_location << n_power) + idx] = W1;
            output[((output_location + 1) << n_power) + idx] = W2;
        }
        else
        {
            if (index2 == 1)
            {
                double angle = M_PI / (v_size << 2);
                Complex64 omega_4n(cos(angle), sin(angle));
                int expo = exponent_calculation(index1, n);
                W1 = omega_4n.inverse();
                W1 = W1.exp(expo);
                W1 = W1 / Complex64(2.0, 0.0);
                W2 = Complex64(0.0, 0.0);
                W3 = W1.negate();
            }

            output[(output_location << n_power) + idx] = W1;
            output[((output_location + 1) << n_power) + idx] = W2;
            output[((output_location + 2) << n_power) + idx] = W3;
        }
    }

    __global__ void E_diagonal_inverse_matrix_mult_single_kernel(
        Complex64* input, Complex64* output, bool last, int n_power)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (last)
        {
            for (int i = 0; i < 2; i++)
            {
                output[idx + (i << n_power)] = input[idx + (i << n_power)];
            }
        }
        else
        {
            for (int i = 0; i < 3; i++)
            {
                output[idx + (i << n_power)] = input[idx + (i << n_power)];
            }
        }
    }

    __global__ void E_diagonal_matrix_mult_kernel(
        Complex64* input, Complex64* output, Complex64* temp, int* diag_index,
        int* input_index, int* output_index, int iteration_count,
        int R_matrix_counter, int output_index_counter, int mul_index,
        bool first1, bool first2, int n_power)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        int offset = first1 ? 2 : 3;
        int L_matrix_loc_ = offset + (3 * mul_index);
        int L_matrix_size = 3;

        int R_matrix_counter_ = R_matrix_counter;
        int output_index_counter_ = output_index_counter;
        int iter_R_m = iteration_count;
        if (first2)
        {
            for (int i = 0; i < iter_R_m; i++)
            {
                int diag_index_ = diag_index[R_matrix_counter_];
                Complex64 R_m = input[idx + (i << n_power)];
                for (int j = 0; j < L_matrix_size; j++)
                {
                    Complex64 L_m =
                        rotated_access(input + ((L_matrix_loc_ + j) << n_power),
                                       diag_index_, idx, n_power);

                    int output_location = output_index[output_index_counter_];

                    Complex64 res = output[(output_location << n_power) + idx];
                    res = res + (L_m * R_m);
                    output[(output_location << n_power) + idx] = res;

                    output_index_counter_++;
                }
                R_matrix_counter_++;
            }
        }
        else
        {
            for (int i = 0; i < iter_R_m; i++)
            {
                int diag_index_ = diag_index[R_matrix_counter_];
                Complex64 R_m =
                    temp[idx +
                         (input_index[R_matrix_counter_ - offset] << n_power)];
                for (int j = 0; j < L_matrix_size; j++)
                {
                    Complex64 L_m =
                        rotated_access(input + ((L_matrix_loc_ + j) << n_power),
                                       diag_index_, idx, n_power);

                    int output_location = output_index[output_index_counter_];

                    Complex64 res = output[(output_location << n_power) + idx];
                    res = res + (L_m * R_m);
                    output[(output_location << n_power) + idx] = res;

                    output_index_counter_++;
                }
                R_matrix_counter_++;
            }
        }
    }

    __global__ void E_diagonal_inverse_matrix_mult_kernel(
        Complex64* input, Complex64* output, Complex64* temp, int* diag_index,
        int* input_index, int* output_index, int iteration_count,
        int R_matrix_counter, int output_index_counter, int mul_index,
        bool first, bool last, int n_power)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        int L_matrix_loc_ = 3 + (3 * mul_index);
        int L_matrix_size = (last) ? 2 : 3;

        int R_matrix_counter_ = R_matrix_counter;
        int output_index_counter_ = output_index_counter;
        int iter_R_m = iteration_count;
        if (first)
        {
            for (int i = 0; i < iter_R_m; i++)
            {
                int diag_index_ = diag_index[R_matrix_counter_];
                Complex64 R_m = input[idx + (i << n_power)];
                for (int j = 0; j < L_matrix_size; j++)
                {
                    Complex64 L_m =
                        rotated_access(input + ((L_matrix_loc_ + j) << n_power),
                                       diag_index_, idx, n_power);

                    int output_location = output_index[output_index_counter_];
                    Complex64 res = output[(output_location << n_power) + idx];
                    res = res + (L_m * R_m);
                    output[(output_location << n_power) + idx] = res;

                    output_index_counter_++;
                }
                R_matrix_counter_++;
            }
        }
        else
        {
            for (int i = 0; i < iter_R_m; i++)
            {
                int diag_index_ = diag_index[R_matrix_counter_];
                Complex64 R_m =
                    temp[idx + (input_index[R_matrix_counter_ - 3] << n_power)];
                for (int j = 0; j < L_matrix_size; j++)
                {
                    Complex64 L_m =
                        rotated_access(input + ((L_matrix_loc_ + j) << n_power),
                                       diag_index_, idx, n_power);

                    int output_location = output_index[output_index_counter_];
                    Complex64 res = output[(output_location << n_power) + idx];
                    res = res + (L_m * R_m);
                    output[(output_location << n_power) + idx] = res;

                    output_index_counter_++;
                }
                R_matrix_counter_++;
            }
        }
    }

    __global__ void vector_rotate_kernel(Complex64* input, Complex64* output,
                                         int rotate_index, int n_power)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        Complex64 rotated = rotated_access(input, rotate_index, idx, n_power);

        output[idx] = rotated;
    }

    // TODO: implement it for multiple RNS prime (currently it only works for
    // single prime)
    __global__ void mod_raise_kernel(Data64* input, Data64* output,
                                     Modulus64* modulus, int n_power)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x; // ring size
        int idy = blockIdx.y; // rns count
        int idz = blockIdx.z; // cipher count

        int location_input = idx + (idz << n_power);
        int location_output =
            idx + (idy << n_power) + ((gridDim.y * idz) << n_power);

        Data64 input_r = input[location_input];
        Data64 result = OPERATOR_GPU_64::reduce_forced(input_r, modulus[idy]);

        output[location_output] = result;
    }

} // namespace heongpu